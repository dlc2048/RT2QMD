#include "hip/hip_runtime.h"

#include "auxiliary.cuh"


namespace deexcitation {


    __device__ float PROJ_M[CHANNEL::CHANNEL_UNKNWON];
    __device__ float PROJ_M2[CHANNEL::CHANNEL_UNKNWON];
    __device__ float PROJ_CB_RHO[CHANNEL::CHANNEL_UNKNWON];

    __constant__ bool BUFFER_HAS_HID;
    __device__ mcutil::RingBuffer* buffer_catalog;

    __device__ hiprandState* rand_state;

    __device__ Nucleus::MassTable* mass_table;

    __device__ Nucleus::LongLivedNucleiTable long_lived_table;

    __device__ float* coulomb_r0;

    __constant__ bool DO_FISSION;

    __constant__ bool USE_DISCRETE_LEVEL;


    __device__ float coulombBarrierRadius(int z, int a) {
        float r = Nucleus::explicitNuclearRadius({ (unsigned char)z, (unsigned char)a });
        if (r <= 0.f) {
            z = min(z, 92);
            r = coulomb_r0[z] * powf((float)a, constants::ONE_OVER_THREE);
        }
        return r;
    }


    __device__ float coulombBarrier(CHANNEL channel, int rz, int ra, float exc_energy) {
        float cb = constants::FP32_FSC_HBARC_MEV * (float)(PROJ_Z[channel] * rz)
            / (coulombBarrierRadius(rz, ra) + PROJ_CB_RHO[channel]);
        if (exc_energy > 0.f)
            cb /= 1.f + sqrtf(exc_energy / (float)ra * 0.5f);
        return cb;
    }


    __host__ hipError_t setBufferHandle(hipDeviceptr_t handle, bool has_hid) {
        M_SOASymbolMapper(mcutil::RingBuffer*, handle, buffer_catalog);
        M_SOAPtrMapper(bool, has_hid, BUFFER_HAS_HID);
        return hipSuccess;
    }


    __host__ hipError_t setPrngHandle(hipDeviceptr_t handle) {
        M_SOASymbolMapper(hiprandState*, handle, rand_state);
        return hipSuccess;
    }


    __host__ hipError_t setMassTableHandle(hipDeviceptr_t handle) {
        M_SOASymbolMapper(Nucleus::MassTable*, handle, mass_table);
        return hipSuccess;
    }


    __host__ hipError_t setStableTable(const Nucleus::LongLivedNucleiTable& table_host) {
        return hipMemcpyToSymbol(HIP_SYMBOL(long_lived_table), &table_host, sizeof(Nucleus::LongLivedNucleiTable));
    }


    __host__ hipError_t setCoulombBarrierRadius(float* cr_arr) {
        M_SOAPtrMapper(float*, cr_arr, coulomb_r0);
        return hipSuccess;
    }


    __host__ hipError_t setEmittedParticleMass(float* mass_arr, float* mass2_arr) {
        hipError_t res;
        res = hipMemcpyToSymbol(HIP_SYMBOL(PROJ_M[0]),  mass_arr,  
            sizeof(float) * CHANNEL::CHANNEL_UNKNWON);
        if (res != hipSuccess) return res;
        res = hipMemcpyToSymbol(HIP_SYMBOL(PROJ_M2[0]), mass2_arr, 
            sizeof(float) * CHANNEL::CHANNEL_UNKNWON);
        if (res != hipSuccess) return res;
        return res;
    }


    __host__ hipError_t setEmittedParticleCBRho(float* rho_arr) {
        return hipMemcpyToSymbol(HIP_SYMBOL(PROJ_CB_RHO[0]), rho_arr,
            sizeof(float) * CHANNEL::CHANNEL_UNKNWON);
    }


    __host__ hipError_t setFissionFlag(bool flag) {
        M_SOAPtrMapper(bool, flag, DO_FISSION);
        return hipSuccess;
    }


}