#include "hip/hip_runtime.h"

#include "channel_evaporation.cuh"
#include "device/shuffle.cuh"

#include <stdio.h>


namespace deexcitation {
    namespace evaporation {


        __device__ float getAlphaParam(CHANNEL channel, int rz, int ra) {
            float alpha = 1.f;
            switch (channel) {
            case CHANNEL::CHANNEL_NEUTRON:
                alpha = getAlphaParamNeutron(ra);
                break;
            case CHANNEL::CHANNEL_PROTON:
                alpha = 1.f + getAlphaParamHydrogen(rz);
                break;
            case CHANNEL::CHANNEL_DEUTERON:
                alpha = 1.f + 0.5f * getAlphaParamHydrogen(rz);
                break;
            case CHANNEL::CHANNEL_TRITON:
                alpha = 1.f + constants::ONE_OVER_THREE * getAlphaParamHydrogen(rz);
                break;
            case CHANNEL::CHANNEL_HELIUM3:
                alpha = 1.f + constants::FOUR_OVER_THREE * getAlphaParamHelium(rz);
                break;
            case CHANNEL::CHANNEL_ALPHA:
                alpha = 1.f + getAlphaParamHelium(rz);
                break;
            default:
                break;
            }
            return alpha;
        }


        __device__ float getBetaParam(CHANNEL channel, int rz, int ra) {
            return channel == CHANNEL::CHANNEL_NEUTRON ? getBetaParamNeutron(ra) : 0.f;
        }


        __device__ float emissionProbability(CHANNEL channel, int z, int a, float mass, float exc_energy) {
            int pz = PROJ_Z[channel];  // emission Z
            int pa = PROJ_A[channel];  // emission A
            int rz = z - pz;           // remnant Z
            int ra = a - pa;           // remnant A

            // allowed channel
            if (ra < pa || ra < rz || rz < 0 || (ra == pa && rz < pz) || (ra > 1 && (ra == rz || rz == 0)))
                return 0.f;

            float delta0 = fission::pairingCorrection(z, a - z);
            if (exc_energy < delta0)
                return 0.f;

            // available kinetic energy
            mass += exc_energy;  // total mass energy
            float res_mass = mass_table[rz].get(ra);

            float eke_max  = 0.5f * ((mass - res_mass) * (mass + res_mass) + PROJ_M2[channel]) / mass - PROJ_M[channel];
            float elim     = coulombBarrier(channel, rz, ra, 0.f);
            if (mass <= res_mass + PROJ_M[channel] + elim)
                return 0.f;

            float eke_min   = 0.f;
            if (elim > 0.f) {
                float res_m = mass - PROJ_M[channel] - elim;
                eke_min = fmaxf(0.f, 0.5f * ((mass - res_m) * (mass + res_m) + PROJ_M2[channel]) / mass - PROJ_M[channel]);
            }

            if (eke_max <= eke_min)
                return 0.f;

            float a0             = getLevelDensityParameter(a);
            float a1             = getLevelDensityParameter(ra);
            float system_entropy = 2.f * sqrtf(a0 * (exc_energy - delta0));
            
            float alpha = getAlphaParam(channel, rz, ra);
            float beta  = getBetaParam(channel, rz, ra);

            float maxea = eke_max * a1;
            float term1 = beta * a1 - 1.5f + maxea;
            float term2 = (2.f * beta * a1 - 3.f) * sqrtf(maxea) + 2.f * maxea;

            float expterm1 = system_entropy <= 160.f ? expf(-system_entropy) : 0.f;
            float expterm2 = 2.f * sqrtf(maxea) - system_entropy;

            expterm2 = expf(fminf(expterm2, 160.f));

            float gfactor = PROJ_S[channel] * alpha * PROJ_M[channel] * PROB_COEFF * powf((float)ra, constants::TWO_OVER_THREE) / (a1 * a1);

            return fmaxf(0.f, (term1 * expterm1 + term2 * expterm2) * gfactor);
        }


        __device__ float emitParticleEnergy(hiprandState* state, CHANNEL channel, int res_z, int res_a, float exc_energy) {
            float mass     = mcutil::cache_univ[CUDA_WARP_SIZE + blockDim.x + threadIdx.x] + exc_energy;
            float res_mass = mass_table[res_z].get(res_a);

            float eke_max  = 0.5f * ((mass - res_mass) * (mass + res_mass) + PROJ_M2[channel]) / mass - PROJ_M[channel];
            
            float elim     = coulombBarrier(channel, res_z, res_a, 0.f);
            if (mass <= res_mass + PROJ_M[channel] + elim)
                return 0.f;
            
            float eke_min = 0.f;
            if (elim > 0.f) {
                float res_m = mass - PROJ_M[channel] - elim;
                eke_min = fmaxf(0.f, 0.5f * ((mass - res_m) * (mass + res_m) + PROJ_M2[channel]) / mass - PROJ_M[channel]);
            }
            
            // rejection unity
            float a1       = getLevelDensityParameter(res_a);
            float xmax     = (sqrtf(0.25f + a1 * eke_max) - 0.5f) / a1;
            float wmax     = xmax * expf(2.f * sqrtf(a1 * (eke_max - xmax)));

            // sample energy
            float x, w;
            do {
                x = (eke_max - eke_min) * (1.f - hiprand_uniform(state));
                w = x * expf(2.f * sqrtf(a1 * (eke_max - x)));
            } while (wmax * hiprand_uniform(state) > w);
            return res_a > 4 ? x + eke_min : eke_max;
        }



        /*
        __device__ float emitParticleEnergy(hiprandState* state, CHANNEL channel, int res_z, int res_a, float exc_energy) {
            //uchar4* cache_zaev = reinterpret_cast<uchar4*>(mcutil::cache_univ + CUDA_WARP_SIZE);
            //uchar4  zaev;

            double mass     = (double)mcutil::cache_univ[CUDA_WARP_SIZE + blockDim.x + threadIdx.x] + (double)exc_energy;
            double res_mass = (double)mass_table[res_z].get(res_a);

            // calculate maximum possible kinetic energy
            double eke_max = 0.5f * ((mass - res_mass) * (mass + res_mass) + PROJ_M2[channel]) / mass - PROJ_M[channel];

            // calculate minimum possible kinetic energy
            double elim    = coulombBarrier(channel, res_z, res_a, 0.f);
            double eke_min = 0.f;
            if (elim > 0.f) {
                double res_m = mass - PROJ_M[channel] - elim;
                eke_min = fmaxf(0.f, 0.5f * ((mass - res_m) * (mass + res_m) + PROJ_M2[channel]) / mass - PROJ_M[channel]);
            }

            // sample energy
            // Geant4 10.x.x method
            double a1   = getLevelDensityParameter(res_a + PROJ_A[channel]);
            double rb   = 4.f * a1 * eke_max;
            rb = sqrt(rb);

            double pex1 = rb < 160.f ? exp(-rb) : 0.f;

            double rk;
            double frk;
            do {
                rk = hiprand_uniform(state);
                rk = 1.f + 1 / rb * log(rk + (1.f - rk) * pex1);
                double q1 = 1.f;
                double q2 = 1.f;
                if (channel == CHANNEL::CHANNEL_NEUTRON) {
                    float beta = (2.12f / (float)res_a / (float)res_a - 0.05f)
                        / (0.76f + 2.2f * powf((float)res_a, -constants::ONE_OVER_THREE));
                    q1 = 1.f + beta / eke_max;
                    q2 = q1 * sqrtf(q1);
                }
                frk = SSQR3 * rk * (q1 - rk * rk) / q2;

            } while (frk < hiprand_uniform(state));

            return eke_max * (1.f - rk * rk) + eke_min;
        }
        */


        __device__ void emitParticle(hiprandState* state, CHANNEL channel) {
            uchar4* cache_zaev = reinterpret_cast<uchar4*>(mcutil::cache_univ + CUDA_WARP_SIZE);
            float   exc_energy = mcutil::cache_univ[CUDA_WARP_SIZE + 3 * blockDim.x + threadIdx.x];

            // update ZA of system
            uchar4  zaev;
            zaev.z = (unsigned char)PROJ_Z[channel];
            zaev.w = (unsigned char)PROJ_A[channel];
            zaev.x = cache_zaev[threadIdx.x].x - zaev.z;
            zaev.y = cache_zaev[threadIdx.x].y - zaev.w;

            assert(cache_zaev[threadIdx.x].x >= zaev.z);
            assert(cache_zaev[threadIdx.x].y >= zaev.w);

            // update cache
            cache_zaev[threadIdx.x] = zaev;

            // momentum of emitted particle
            float eke_emit  = emitParticleEnergy(state, channel, (int)zaev.x, (int)zaev.y, exc_energy);

            float mass_emit = PROJ_M[channel];
            float momentum  = eke_emit * (eke_emit + 2.f * mass_emit);  // now it is the square of norm

            // calculate the excitation energy of primary remnant (use double to avoid FP error) 
            float  mass_rem = mass_table[zaev.x].get(zaev.y);
            float exc_rem  =
                + mcutil::cache_univ[CUDA_WARP_SIZE + blockDim.x + threadIdx.x]  // primary mass
                + exc_energy  // primary excitation
                - mass_emit   // emitted mass
                - eke_emit;   // kinetic energy of emitted particle
            exc_rem = sqrt((exc_rem * exc_rem - momentum)) - mass_rem;
            assert(exc_rem > -0.1f);
            exc_rem = fmaxf(exc_rem, 0.f);

            // update cache
            mcutil::cache_univ[CUDA_WARP_SIZE + 1 * blockDim.x + threadIdx.x] = mass_rem;
            mcutil::cache_univ[CUDA_WARP_SIZE + 2 * blockDim.x + threadIdx.x] = mass_emit;
            mcutil::cache_univ[CUDA_WARP_SIZE + 3 * blockDim.x + threadIdx.x] = exc_rem;
            mcutil::cache_univ[CUDA_WARP_SIZE + 4 * blockDim.x + threadIdx.x] = 0.f;

            // random isotropic direction
            float cost, sint;
            float cosp, sinp;
            float angle;

            // now momentum is norm
            momentum = sqrtf(momentum);

            // polar
            cost  = 1.f - 2.f * hiprand_uniform(state);
            sint  = sqrtf(fmaxf(0.f, 1.f - cost * cost));
            // azimuthal
            angle = constants::FP32_TWO_PI * hiprand_uniform(state);
            __sincosf(angle, &sinp, &cosp);

            mcutil::cache_univ[CUDA_WARP_SIZE + 5 * blockDim.x + threadIdx.x] = momentum * sint * cosp;  // X
            mcutil::cache_univ[CUDA_WARP_SIZE + 6 * blockDim.x + threadIdx.x] = momentum * sint * sinp;  // Y
            mcutil::cache_univ[CUDA_WARP_SIZE + 7 * blockDim.x + threadIdx.x] = momentum * cost;         // Z

            // Momentum of primary remnant is (-px, -py, -pz)
        }


    }
}